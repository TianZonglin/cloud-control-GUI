#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "include/cudawrapper.h"
#include <hiprand/hiprand_kernel.h>
#include "include/gpubundling.h"


texture<float,hipTextureType2D,hipReadModeElementType>		 texDensity;						//State var:  the float 2D density texture
texture<unsigned int,hipTextureType1D,hipReadModeElementType>	 texCount;							//!!
texture<BundlingPoint,hipTextureType1D,hipReadModeElementType> texSites;							//State var:  the float2 1D sites vector
texture<int,hipTextureType1D,hipReadModeElementType>			 texStarts;							//State var:
texture<int,hipTextureType1D,hipReadModeElementType>			 texEdges;							//State var:  start-offsets of all edges in texSites[]
texture<float,hipTextureType1D,hipReadModeElementType>		 texEdgeProfile;					//State var:  the edge profile (controlling the advection along an edge)
texture<unsigned int,hipTextureType2D,hipReadModeElementType>	 texInt2D;							//!!



__constant__ float												 c_Kernel[MAX_KERNEL_LENGTH];		//State var:  the kernel data (stored as constant for speed)
__constant__ int kernel_radius;																		//State var:  the current kernel radius
__constant__ int imageW,imageH;																		//State vars: sizes of the image used allover through the code
__device__   int numControlPoints;




//--- GPU-specific defines -----------------------------------------------------------------

#define IMAD(a, b, c) ( __mul24((a), (b)) + (c) )			//Maps to a single instruction on G8x / G9x / G10x

inline int iDivUp(int a, int b)								//Round a / b to nearest higher integer value
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

inline int iAlignUp(int a, int b)							//Align a to nearest higher multiple of b
{
    return (a % b != 0) ?  (a - a % b + b) : a;
}



__global__ void convolutionRowsKernel(float* output)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    if (ix >= imageW || iy >= imageH) return;							//Careful not to index outside image
																		//REMARK: For advection, we don't actually need the density field over the entire image,
    const float  x = (float)ix + 0.5f;									//		  but only over a 2x2 nbhood of the sampling points (for gradient computation).
    const float  y = (float)iy + 0.5f;									//		  However, for shading, we need it over all pixels covered by edges. Since it's
    float sum = 0;														//		  hard to limit computation there, we compute density over the entire image.

    /*
    bool close2site = false;
    for(short X = x-1; X <= x+1 && !close2site; ++X)
    for(short Y = y-1; Y <= y+1 && !close2site; ++Y)
    {
    if (tex2D(texDensity,X,Y))
    close2site = true;
    }

    if (close2site)
*/
	for(short k = -kernel_radius; k <= kernel_radius; ++k)
	sum += tex2D(texDensity,x+k,y)*c_Kernel[kernel_radius-k];

    output[IMAD(iy,imageW,ix)] = sum;
}


__global__ void convolutionColumnsKernel(float* output)
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    if (ix >= imageW || iy >= imageH) return;							//Careful not to index outside image

    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;
    float sum = 0;

/*
    bool close2site = false;
    for(short X = x-1; X <= x+1 && !close2site; ++X)
    for(short Y = y-1; Y <= y+1 && !close2site; ++Y)
    {
        if (tex2D(texDensity,X,Y))
           close2site = true;
    }

    if (close2site)
*/
	for(short k = -kernel_radius; k <= kernel_radius; ++k)
		sum += tex2D(texDensity,x,y+k)*c_Kernel[kernel_radius-k];

	output[IMAD(iy,imageW,ix)] = sum;
}

void convolutionRowsColumns(float* output,hipArray *a_Src,int imageW,int imageH,bool rows_or_columns)
{
    dim3 threads(NTHREADS_X,NTHREADS_Y);
    dim3 blocks(iDivUp(imageW,threads.x),iDivUp(imageH,threads.y));

    hipBindTextureToArray(texDensity,a_Src);
	if (rows_or_columns)
		convolutionRowsKernel<<<blocks,threads>>>(output);
	else
		convolutionColumnsKernel<<<blocks,threads>>>(output);
	hipUnbindTexture(texDensity);

    hipDeviceSynchronize();
}


extern "C" void convolutionGPU(float* d_Output,hipArray* a_Src,int imageW,int imageH)
{
	//1. Convolve on rows (a_Src -> d_Output)
	hipDeviceSynchronize();

	convolutionRowsColumns(d_Output,a_Src,imageW,imageH,true);

    //2. Copy row-convolution to texture (d_Output -> a_Src)
    //   While CUDA kernels can't write to textures directly, this copy is inevitable
	hipMemcpyToArray(a_Src,0,0,d_Output,imageW*imageH*sizeof(float),hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

    //3. Convolve on columns (a_Src -> d_Output)
	convolutionRowsColumns(d_Output,a_Src,imageW,imageH,false);
}





__global__ void kernelSiteInitCount(unsigned int* output,int numpts)		//Init the image output[] with the #sites at ech pixel
{
    int offs = IMAD(blockIdx.x, blockDim.x, threadIdx.x);

	if (offs < numpts)														//careful not to index outside the site-vector..
	{
	  const BundlingPoint p = tex1Dfetch(texSites,offs);					//find coords of current site
	  if (p.x<0) return;													//this is a marker: nothing to do

	  int site_offs = IMAD(int(p.y),imageW,int(p.x));						//Increment site-count for pixel under it
	  atomicAdd(output+site_offs,1);										//REMARK: This seems expensive, but it only occurs when sites DO overlap...
	}
}

__global__ void kernelSiteInitFloat(float* output,int numpts,float value)	//Init the image output[] with 'value' at all site locations
{
	int offs = IMAD(blockIdx.x,blockDim.x,threadIdx.x);

	if (offs < numpts)
	{
	  const BundlingPoint p  = tex1Dfetch(texSites,offs);					//find coords of current site
	  if (p.x<0) return;													//this is a marker: nothing to do

	  int site_offs = IMAD(int(p.y),imageW,int(p.x));
output[site_offs] += p.w; //!!!value;											//Set pixel under site to 'value'
	}																		//WARNING: This underestimates density where multiple sites fall on the same pixel
																			//due to the fact that += is not atomic on threads that want to write to the same pixel
}

__global__ void kernelSiteInitCount2Float(float* output,int imageW)			//Copy count image (from texCount) to floating-point image output[]
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    if (ix >= imageW || iy >= imageH) return;								//Careful not to index outside image

	const int offs = IMAD(iy,imageW,ix);
    output[offs] = tex1Dfetch(texCount,offs);
}







extern "C" void initializeSiteLocations(CPUBundling::DENSITY_ESTIM dens_estim,hipArray* a_Src,float* d_Output,unsigned int* d_Count,BundlingPoint* d_points,int npoints,float value,int imageW,int imageH)
{
    hipMemcpyToSymbol(HIP_SYMBOL(::imageW), &imageW, sizeof(int));					//Store these state-vars in const memory, for fast access later
    hipMemcpyToSymbol(HIP_SYMBOL(::imageH), &imageH, sizeof(int));

	int threads  = NTHREADS;											//Prepare the site-init kernel: this reads a vector of 2D sites
	int numpts_b = iAlignUp(npoints,threads);							//Find higher multiple of blocksize than # sites
	int blocks   = numpts_b/threads;

	hipBindTexture(0,texSites,d_points);								//Bind 2D sites to a 1D texture

	if (dens_estim==CPUBundling::DENSITY_FAST)							//Fast density estimation: simply record if there's no or one..more sites/pixel
	{
		hipMemset(d_Output,0,sizeof(float)*imageW*imageH);				 //Zero the density texture
		kernelSiteInitFloat<<<blocks,threads>>>(d_Output,npoints,value); //Set all site-locations in the density map to 'value'
	}
	else	//DENSITY_EXACT												//Exact density estimation, pass 1
	{																	//Count #sites/pixel (slower, uses atomic ops)
		hipMemset(d_Count,0,sizeof(unsigned int)*imageW*imageH);
		kernelSiteInitCount<<<blocks,threads>>>(d_Count,npoints);		//Count #sites/pixel in d_Count
	}

	hipDeviceSynchronize();
	hipUnbindTexture(texSites);										//Done with the sites

	if (dens_estim==CPUBundling::DENSITY_EXACT)							//Exact density estimation, pass 2
	{
		dim3 threads(NTHREADS_X,NTHREADS_Y);
		dim3 blocks(iDivUp(imageW,threads.x),iDivUp(imageH,threads.y));
		hipBindTexture(0,texCount,d_Count);
		kernelSiteInitCount2Float<<<blocks,threads>>>(d_Output,imageW);	//Simply copy d_Count to the floating-point d_Output
		hipDeviceSynchronize();
		hipUnbindTexture(texCount);
	}

	hipMemcpyToArray(a_Src,0,0,d_Output,imageW*imageH*sizeof(float),hipMemcpyDeviceToDevice);


    texDensity.addressMode[0] = hipAddressModeClamp;                   //Not sure if this is needed, but I want to make sure that
    texDensity.addressMode[1] = hipAddressModeClamp;                   //texDensity is bilinearly interpolated when read at floating-point locations
    texDensity.filterMode     = hipFilterModeLinear;
    texDensity.normalized     = false;
}






__global__ void kernelSiteInitDirs(unsigned int* output, int numpts)		//Init the image output[] with the dirs at ech pixel
{
    int offs = IMAD(blockIdx.x, blockDim.x, threadIdx.x);

	if (offs < numpts)														//careful not to index outside the site-vector..
	{
	  BundlingPoint p = tex1Dfetch(texSites,offs);							//find current site
	  if (p.x<0) return;													//this is a marker: don't write it into the texture

	  unsigned int site_offs = IMAD(imageW,int(p.y),int(p.x));				//Find pixel under site
	  output[site_offs] = offs;												//Write its point-ID into the texture
	}
}



extern "C" void initializeSiteMap(unsigned int* siteMap,BundlingPoint* d_points,int npoints,int imageW,int imageH)
{
    hipMemcpyToSymbol(HIP_SYMBOL(::imageW), &imageW, sizeof(int));					//Store these state-vars in const memory, for fast access later
    hipMemcpyToSymbol(HIP_SYMBOL(::imageH), &imageH, sizeof(int));

	int threads  = NTHREADS;											//Prepare the site-init kernel: this reads a vector of 2D sites
	int numpts_b = iAlignUp(npoints,threads);							//Find higher multiple of blocksize than # sites
	int blocks   = numpts_b/threads;

	hipBindTexture(0,texSites,d_points);								//Bind 2D sites to a 1D texture

																		//Set the site-IDs as values of their pixel locations in texture
	hipMemset(siteMap,0,sizeof(unsigned int)*imageW*imageH);			//First, zero up the sitemap-texture
	kernelSiteInitDirs<<<blocks,threads>>>(siteMap,npoints);			//Next, write sites in texture
	hipDeviceSynchronize();
	hipUnbindTexture(texSites);										//Done with the site-texture
}



__device__ inline float2 edgeTangent(int offs)
{
	BundlingPoint p   = tex1Dfetch(texSites,offs);						//Get current point to advect
	BundlingPoint q   = tex1Dfetch(texSites,offs+1);					//Get current point to advect
	if (q.x<0)
	{
	   q = p;
	   p = tex1Dfetch(texSites,offs-1);
	}

	float eps = 1.0e-5;
	float2 tv = make_float2(q.x-p.x,q.y-p.y);
	float tvn = rsqrtf(tv.x*tv.x+tv.y*tv.y+eps);
	tv.x *= tvn;														//tv: tangent vector to edge at p
	tv.y *= tvn;

	return tv;
}




//REMARK: rho, in the dir-advection, cannot always be 1 - its length must depend on compatibility
//REMARK: we have 2 possibilities for dir-compatibility: initial dirs, or dirs of the bundled edges

//REMARK: The directional factor should be a function of
//		  -the length of the attracted edge: longer gets more dir-influence
//		  -the length of the attracting edge: ???

//REMARK: Modify ::draw (shading) to ignore shading if edge-dir (as given by tangent) is very different than grad density
//
//




__device__ float interpAngle(float a,float b,float blend)
{
	float ix = cosf(a);
	float iy = sinf(a);
	float jx = cosf(b);
	float jy = sinf(b);

	float x = (1-blend)*ix + blend*jx;
	float y = (1-blend)*iy + blend*jy;
	float n = rsqrtf(x*x+y*y);
	x *= n;
	y *= n;

	float res = atan2f(y,x);
	if (res<0) res += 2*M_PI;
	return res;
}


__device__ float dir2angle(const float2& d)
{
	float a = acosf(d.x);
	if (d.y<0) a += M_PI;
	return a;
}


__device__ float2 dirDensityGradient(int offs,int imageW,float rep_strength)
{																		//Compute density-gradient for advection that considers edge-directions
	const float        eps = 1.0e-4;
	const BundlingPoint  p = tex1Dfetch(texSites,offs);					//get current site, where we want to compute the dir. density
	float              a_p = p.w;										//get edge-direction at site 'p' (angle in [0,2*M_PI])

	float2 grad = make_float2(0,0);										//accumulates compatible tangent-vectors around p
	int minX  = fmaxf(0.0f,p.x-kernel_radius), maxX = fminf(float(imageW),p.x+kernel_radius);
	int minY  = fmaxf(0.0f,p.y-kernel_radius), maxY = fminf(float(imageW),p.y+kernel_radius);
	float r2    = kernel_radius*kernel_radius;

	int N=0;															//counts #sites that we gather from
	float comps = 0;
	for(int i=minX;i<=maxX;++i)											//scan circle of radius 'radius_kernel' around p:
	{																	//(this gathering step is SLOW)
	  int dx2 = (p.x-i)*(p.x-i);
	  for(int j=minY;j<=maxY;++j)
	  {
		  int d2 = dx2+(p.y-j)*(p.y-j);
		  if (d2>r2) continue;											//ignore points outside kernel_radius

		  unsigned int q = tex2D(texInt2D,i,j);							//get possible site at pixel (i,j)
		  if (q==0) continue;											//no site there? nothing to gather at p from (i,j)

		  float  a_q = tex1Dfetch(texSites,q).w;						//get edge-direction at site 'q' (angle in [0,2*M_PI])

		  float comp = fabsf(a_q-a_p);									//compute compatibility of site 'q' with site 'p'
		  if (comp>M_PI) comp = __fmaf_rz(2,M_PI,-comp);				//make comp reside in [0,pi]
		  comp = __fmaf_rz(-2/M_PI,comp,1);								//comp in [-1,1]: comp=1=parallel, comp=0=ortho, comp=-1=antiparallel
																		//hence: parallel edges attract themselves, antiparallel ones repel themselves

		  if (comp<0) comp *= rep_strength; 							//restrict repulsion

		  float2 g = make_float2(i-p.x,j-p.y);							//compute normalized density gradient due to site (i,j)
		  float gn = comp*__expf(-4*d2/r2);
		  g.x *= gn; g.y *= gn;

		  grad.x += g.x;												//accumulate gradient weighted by directional compatibility
		  grad.y += g.y;
		  comps += fabsf(comp);											//accumulate weights of summed gradients
		  ++N;
	  }
   }

   float r = grad.x*grad.x+grad.y*grad.y;								//normalize the resulting gradient
   if (r<1.0e-4) { grad.x = grad.y = 0; }
   else
   {
       float K = comps / N;
	   float rn = K*rsqrtf(r+eps);
	   grad.x *= rn; grad.y *= rn;
   }

   return grad;
}




__global__ void kernelAdvectSites(BundlingPoint* output,float h,float numpts)
{																			//Advect graph in density-gradient
    int offs = IMAD(blockIdx.x, blockDim.x, threadIdx.x);

	if (offs<numpts)
	{
		BundlingPoint p   = tex1Dfetch(texSites,offs);						//Get current point to advect

		if (p.x<0)															//Marker: copy it, don't advect it
		{
		    output[offs] = p;
		}
		else																//Regular point: advect it
		{
			float  v_d = tex2D(texDensity,p.x,p.y-1);						//Get density at that point and at its nbs
			float  v_l = tex2D(texDensity,p.x-1,p.y);
			float  v_r = tex2D(texDensity,p.x+1,p.y);
			float  v_t = tex2D(texDensity,p.x,p.y+1);
			BundlingPoint g = make_float4(v_r-v_l,v_t-v_d,p.z,p.w);			//Compute density gradient, simple forward difference method

			const float eps = 1.0e-4;										//Ensures we don't next get div by 0 for 0-length vectors
			float gn = g.x*g.x+g.y*g.y;
			if (gn<eps) gn = 0;
			else gn = rsqrtf(gn);											//Robustly normalize the gradient

			float  k = h*p.z*gn;                                            //k = displacement of current point 'p'
			g.x *= k; g.x += p.x;											//Advect current point
			g.y *= k; g.y += p.y;

      output[offs] = g;												//Write displaced point to 'output'

		}
	}
}









__global__ void kernelAdvectSitesDirectional(BundlingPoint* output,float h,float numpts,int imageW,float rep_strength)
{																			//Advect graph in density-gradient caused only by directionally-compatible sites
    int offs = IMAD(blockIdx.x, blockDim.x, threadIdx.x);

	if (offs<numpts)
	{
		BundlingPoint p   = tex1Dfetch(texSites,offs);						//Get current point to advect

		if (p.x<0)															//Marker: copy it, don't advect it
		{
		    output[offs] = p;
		}
		else																//Regular point: advect it
		{
			float2 grad = dirDensityGradient(offs,imageW,rep_strength);		//Compute directional density gradient at current point
			BundlingPoint g = make_float4(grad.x,grad.y,p.z,p.w);

			float  k = h*p.z;
			g.x *= k; g.x += p.x;											//Advect current point
			g.y *= k; g.y += p.y;

			output[offs] = g;												//Write displaced point to 'output'
		}
	}
}





extern "C" void advectSites(BundlingPoint* out_points,BundlingPoint* in_points,int npoints,float* a_Src,unsigned int* d_siteMap,int imageW,int imageH,
							float h,bool tangent,float rep_strength)
{																		//Advect the sites, one step, along its density gradient
	dim3 block = dim3(NTHREADS);										//Prepare the site-init kernel: this reads a vector of 2D sites
	int numpts_b = iAlignUp(npoints,block.x);							//Find higher multiple of blocksize than # sites
	dim3 grid  = dim3(numpts_b/block.x);								//Number of blocks, each of 'block' threads. Each thread advects a point.

	hipBindTexture(0,texSites,in_points);								//Bind 2D sites to a 1D texture (for reading)

    if (!tangent)														//No directional bundling:
	{
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
		hipBindTexture2D(0,texDensity,a_Src,channelDesc,imageW,imageH,4*imageW);
																		//Bind density to a 2D texture (for reading)
		kernelAdvectSites<<<grid,block>>>(out_points,h,npoints);		//Advect the sites from in_points to out_points	in the density gradient
		hipDeviceSynchronize();
		hipUnbindTexture(texDensity);
	}
	else																//Directional bundling:
	{
		texInt2D.filterMode = hipFilterModePoint;						//Bind side-ID image to a 2D texture (for reading)
		texInt2D.normalized = false;
		hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindUnsigned);
		hipBindTexture2D(0,texInt2D,d_siteMap,channelDesc2,imageW,imageH,4*imageW);

		kernelAdvectSitesDirectional<<<grid,block>>>(out_points,h,npoints,imageW,rep_strength);
		hipDeviceSynchronize();										//Advect the sites from in_points to out_points	in the density gradient
		hipUnbindTexture(texInt2D);
	}

	hipUnbindTexture(texSites);										//Done with the textures
}



__global__ void kernelSmoothLines(BundlingPoint* out_points,int npoints,float t,int L)
{
    int offs = IMAD(blockIdx.x, blockDim.x, threadIdx.x);

	if (offs>=npoints) return;												//Care not to index outside the site array

	out_points += offs;

	BundlingPoint crtp = tex1Dfetch(texSites,offs);							//Is current point an end-of-line marker?
	if (crtp.x<0)															//If so, copy it w/o smoothing
	{
	   *out_points = crtp;
	}
	else																	//Smooth current point w.r.t. a window of [-L,L] points centered at it:
	{
	   float2 pc = make_float2(0,0);
	   unsigned char pcount = 0;

	   int km = (offs>L)? offs-L : 0;										//Make sure we don't pass left to 0-th point..
	   #pragma unroll
	   for(int k=offs-1;k>=km;--k)											//Gather points 'upstream' until end-of-kernel or beginning-of-line:
	   {
		  BundlingPoint pinp = tex1Dfetch(texSites,k);
		  if (pinp.x<0) break;												//Stop at line-beginning

		  pc.x += pinp.x;
		  pc.y += pinp.y;
		  ++pcount;
	   }

	   #pragma unroll
	   for(int kM=offs+L;offs<=kM;++offs)									//Gather points 'downstream' until end-of-kernel or end-of-line:
	   {
		  BundlingPoint pinp = tex1Dfetch(texSites,offs);
		  if (pinp.x<0) break;												//Stop at line-end

		  pc.x += pinp.x;
		  pc.y += pinp.y;
		  ++pcount;
	   }

	   t *= crtp.z;

	   const float k = t/pcount;											//Linear interpolation between point and average of its neighbors
	   crtp.x *= 1-t;
	   crtp.x += pc.x*k;
	   crtp.y *= 1-t;
	   crtp.y += pc.y*k;
	   *out_points = crtp;
	}
}



extern "C" void smoothLines(BundlingPoint* out_points,BundlingPoint* in_points,int npoints,float t,float h,float filter_kernel,int niter)	//Laplacian smoothing of graph-drawing edges
{
	dim3 threads = dim3(NTHREADS);										//Prepare the smooth kernel: this reads a vector of 2D sites
	int numpts_b = iAlignUp(npoints,threads.x);							//Find npoints upper-rounded to a multiple of block.x
	dim3 blocks  = dim3(numpts_b/threads.x);							//Find #blocks fitting numpts_b


	const int L = int(filter_kernel/h);									//Compute 1D Laplacian filter size, in #points, which corresponds to 'filter_kernel' space units
	if (L==0) return;													//Don't do smoothing if filter-size is zero..

	BundlingPoint *out = out_points, *inp = in_points;
	for(int i=0;i<niter;++i)											//Perform several Laplacian iterations:
	{
		hipBindTexture(0,texSites,inp);								//Bind 2D sites to a 1D texture (for reading), unbinds any possibly-bound texture
		kernelSmoothLines<<<blocks,threads>>>(out,npoints,t,L);
		hipDeviceSynchronize();

		BundlingPoint* tmp = out;										//Swap input vs output for next iteration
		out = inp;
		inp = tmp;
	}
	hipUnbindTexture(texSites);										//Done with the texture
}











extern "C" void setConvolutionKernel(float* h_Kernel,int sz,int rad)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, sz * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel_radius), &rad, sizeof(int));
}


__global__ void kernelComputeShading(float* output,int imageW,int imageH,float lx,float ly,float lz,float h_max)
{																			//Compute shading from density gradient
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    if (ix >= imageW-1 || iy >= imageH-1) return;							//Careful not to index outside image

    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

	const float K = imageW/h_max;

	float v  = tex2D(texDensity,x,y);										//Compute normal to density graph from density gradient
	float vx = tex2D(texDensity,x+1,y);										//Normal to surface is the vector (drho/dx,drho/dy,-1)
	float vy = tex2D(texDensity,x,y+1);
	float gx = (vx-v)*K;
	float gy = (vy-v)*K;
	float g2 = gx*gx+gy*gy+1;
	float gn = rsqrtf(g2);													//Normalize the surface-normal

	int offs = IMAD(iy,imageW,ix);

	float shade = (gx*gn*lx+gy*gn*ly-gn*lz);								//Shading = dot-product of normal with light
	if (shade<0) shade=0;
    output[offs] = shade;
}





__global__ void kernelComputeShadingNorm(float* output,int imageW,int imageH,float lx,float ly,float lz,float R)
{																			//Compute shading from gradient of locally-normalized density
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    if (ix >= imageW-1 || iy >= imageH-1) return;							//Careful not to index outside image

	float v_max = 0;
	const float R2 = R*R;
	for(int j=iy-R;j<=iy+R;++j)												//Determine the local density max in a
	for(int i=ix-R;i<=ix+R;++i)												//radius R around current point
	{																		//We'll use this to locally normalize the density in [0,1]
		float r2 = (i-ix)*(i-ix)+(j-iy)*(j-iy);								//This constrains the local gradient to decent ranges
		if (r2>R2) continue;
		float v  = tex2D(texDensity,i+0.5f,j+0.5f);							//(so we can next compute a nice gradient and thus shading)
		if (v>v_max) v_max = v;
	}

    if (v_max==0) v_max = 1;                                                //Careful not to divide by zero.

	v_max /= R;																//We normalize the density in [0,1]. The range of x and y is (imageW,imageH).
																			//To get a proportionate shape, whose normal is useful for shading, we
																			//scale the density in [0,R], corresponding to the height of a 'tube'
																			//of radius R. See also below how v_max is used.
    const float x = ix + 0.5f;
    const float y = iy + 0.5f;
	float v  = tex2D(texDensity,x,y);										//Compute normal to density graph from density gradient

	float vx = tex2D(texDensity,x+1,y);										//Normal to surface is the vector (drho/dx,drho/dy,-1)
	float vy = tex2D(texDensity,x,y+1);
	float gx = (vx-v)/v_max;												//gx,gy = partial derivatives of density w.r.t. x,y
	float gy = (vy-v)/v_max;
	float gl = gx*gx+gy*gy+1;
	float gn = rsqrtf(gl);													//Normalize the surface-normal

	float shade = (gx*gn*lx+gy*gn*ly-gn*lz);								//Shading = dot-product of normal with light
	if (shade<0) shade=0;

	int offs = IMAD(iy,imageW,ix);
    output[offs] = shade;
}



__global__ void kernelComputeShadingTube(float* output,int imageW,int imageH,float R)
{																			//Compute pseudo-shading from density. Take care, this is NOT a physically correct
                                                                            //shading. It is simply the locally normalized density, interpreted as luminance.

//WARNING:  This function is not fully correct. The issue is that R varies continuously (from the caller), since it's a float.
//          But the sampling of texDensity is done at increments of +/- 1 pixel. So, small changes of R may not be 'felt' by this code
//          when computing v_max (of texDensity) within a kernel of radius R. As such, when smoothly varying R in the caller,
//          the shading computed by this (output[]) will exhibit some sharp jumps when R goes over integer bounds (like from 3.9 to 4.1).
//


    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    if (ix >= imageW-1 || iy >= imageH-1) return;							//Careful not to index outside image

	float v_max = 0;
	const float R2 = R*R;
	for(int j=iy-R;j<=iy+R;++j)												//Determine the local density max in a
	for(int i=ix-R;i<=ix+R;++i)												//radius R around current point
	{																		//We'll use this to locally normalize the density in [0,1]
		float r2 = (i-ix)*(i-ix)+(j-iy)*(j-iy);
		if (r2>R2) continue;
		float v  = tex2D(texDensity,i+0.5f,j+0.5f);
		if (v>v_max) v_max = v;
	}

    if (v_max==0) v_max=1;                                                  //Avoid division by zero

	float v  = tex2D(texDensity,ix,iy);                                     //Compute density locally normalized in [0,1]
	int offs = IMAD(iy,imageW,ix);
    output[offs] = v/v_max;                                                 //This is the normalized density at the current point.
                                                                            //More pronounced tube effects can be gotten by raising this to pow(2)
}





__global__ void kernelComputeDT(float* output,int imageW,int imageH)		//Compute shading from density gradient
{
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    if (ix >= imageW-1 || iy >= imageH-1) return;							//Careful not to index outside image

	const float v_min = 0.1;

	const int R = 10;
	int d2min = R*R;
	for(int i=ix-R;i<=ix+R;++i)
	for(int j=iy-R;j<=iy+R;++j)
	{
		const float  x = (float)i + 0.5f;
		const float  y = (float)j + 0.5f;
		float v  = tex2D(texDensity,x,y);										//Compute normal to density graph from density gradient
		if (v<v_min)
		{
			float d2 = (i-ix)*(i-ix)+(j-iy)*(j-iy);
			if (d2<d2min) d2min=d2;
		}
	}

	int offs = IMAD(iy,imageW,ix);
    output[offs] = sqrtf(d2min);
}



extern "C" void computeDT(float* d_Output,hipArray* a_Src,int imageW,int imageH)
{
    dim3 threads(NTHREADS_X,NTHREADS_Y);
    dim3 blocks(iDivUp(imageW,threads.x),iDivUp(imageH,threads.y));

	hipMemcpyToArray(a_Src,0,0,d_Output,imageW*imageH*sizeof(float),hipMemcpyDeviceToDevice);		//Copy density to a_Src, since we'll overwrite it

	hipBindTextureToArray(texDensity,a_Src);														//Bind density to a 2D texture (for reading)

	kernelComputeDT<<<blocks,threads>>>(d_Output,imageW,imageH);									//Compute DT from density
	hipDeviceSynchronize();

	hipUnbindTexture(texDensity);
}





extern "C" void computeShading(float* d_Output,hipArray* a_Src,int imageW,int imageH,const Point3d& light,float radius,bool tube_style)
{
    dim3 threads(NTHREADS_X,NTHREADS_Y);
    dim3 blocks(iDivUp(imageW,threads.x),iDivUp(imageH,threads.y));

	hipMemcpyToArray(a_Src,0,0,d_Output,imageW*imageH*sizeof(float),hipMemcpyDeviceToDevice);			//Copy density to a_Src, since we'll overwrite it


	hipBindTextureToArray(texDensity,a_Src);															//Bind density to a 2D texture (for reading)

	if (tube_style)
	    kernelComputeShadingTube<<<blocks,threads>>>(d_Output,imageW,imageH,radius);						  //Compute tube-like shading
    else
	    kernelComputeShadingNorm<<<blocks,threads>>>(d_Output,imageW,imageH,light.x,light.y,light.z,radius);  //Compute shading from density gradient
	hipDeviceSynchronize();

	hipUnbindTexture(texDensity);
}



//------------------------------


__device__ inline float dist(const BundlingPoint& p,const BundlingPoint& q)
{
	return sqrtf((p.x-q.x)*(p.x-q.x)+(p.y-q.y)*(p.y-q.y));
}



__global__ void kernelResampleCount(int* e_count,int n_edges,float delta)
{
        int e_idx = IMAD(blockIdx.x, blockDim.x, threadIdx.x);			//Get edge number
	if (e_idx>=n_edges) return;						//Careful not to index outside edge-vector


	int	        i    = tex1Dfetch(texStarts,e_idx);			//Find 1st edge-point in texSites
	BundlingPoint   prev = tex1Dfetch(texSites,i);				//First point on current edge
	float		crtd = delta;
	int	        n_newp = 1;						//Add 1st point of input polyline to resampled one
	++i;


	BundlingPoint crt = tex1Dfetch(texSites,i);
	for(;;)									//resample input polyline:
	{
		float newdist = dist(crt,prev);					//distance from last resampled point to i-th input point

		if (newdist<crtd)						//i-th input point closer to 'prev' than remaining fraction of delta:
		{								//skip i-th point
			crtd -= newdist;
			prev  = crt;
			++i;
			crt = tex1Dfetch(texSites,i);
			if (crt.x<0) break;
		}
		else								//i-th input point farther from 'prev' than remaining fraction of delta:
		{
			float t = crtd/newdist;
			prev.x = prev.x*(1-t) + crt.x*t;			//add new point to resampling
			prev.y = prev.y*(1-t) + crt.y*t;			//add new point to resampling
			++n_newp;
			crtd = delta;						//reset delta to whatever we want to achieve
		}
	}

	if (crtd<delta) ++n_newp;
	++n_newp;								//include marker in #points to be stored for this edge
  if(n_newp == 2) ++n_newp;

	e_count[e_idx] = n_newp;						//save #points/edge to generate after resampling
}



__global__ void kernelResample(BundlingPoint* new_pts,int n_edges,float delta,float jitter,hiprandState* state)
{

        int e_idx = IMAD(blockIdx.x, blockDim.x, threadIdx.x);			//Get edge number
	if (e_idx>=n_edges) return;						//Careful not to index outside edge-vector


	int			i    = tex1Dfetch(texStarts,e_idx);		//Find 1st edge-point in texSites
	BundlingPoint           prev = tex1Dfetch(texSites,i);			//First point on current edge

	float		        crtd = delta;
	int			n_newp = tex1Dfetch(texEdges,e_idx);
	int			inxt = tex1Dfetch(texEdges,e_idx+1);
	int			NP   = inxt-n_newp-2;
	if (NP<1)		NP=1;

	const int		EPSZ = EDGE_PROFILE_SIZE-1;

	int omp = 0;

	new_pts += n_newp;							//Here will the resampled edge's points be placed

	*new_pts++ = prev;							//add 1st point of input polyline to resampled one
	++i;

	++omp;

	hiprandState lState = state[threadIdx.x];				//cache random generator for speed, since we'll modify it locally

	BundlingPoint crt = tex1Dfetch(texSites,i);
	for(int j=1;;)								//resample input polyline:
	{
		float newdist = dist(crt,prev);					//distance from last resampled point to i-th input point

		if (newdist<crtd)						//i-th input point closer to 'prev' than remaining fraction of delta:
		{								//skip i-th point
			crtd -= newdist;
			prev  = crt;
			++i;
			crt = tex1Dfetch(texSites,i);
			if (crt.x<0) break;
		}
		else								//i-th input point farther from 'prev' than remaining fraction of delta:
		{
			float t  = crtd/newdist;
			float r  = hiprand_uniform(&lState)*2-1;			//generate random number in [-1..1]
			float rt = t*(1+r*jitter);				//jitter currently-generated point
			BundlingPoint rp;
			rp.x   = prev.x*(1-rt) + crt.x*rt;
			rp.y   = prev.y*(1-rt) + crt.y*rt;

			int pidx = int(j*EPSZ/NP);
			rp.z = tex1Dfetch(texEdgeProfile,pidx);			//apply edge profile on newly, resampled, edge
			rp.w = crt.w; //!!!interpAngle(prev.w,crt.w,rt);

			*new_pts++ = rp;					//add new resampled point to output

			++omp;

			if (omp==NP) break;

			prev.x = prev.x*(1-t) + crt.x*t;			//keep NON-jittered point as next point;
			prev.y = prev.y*(1-t) + crt.y*t;			//this ensures we get here EXACTLY the same polyline sampling as in kernelResampleCount()

			crtd = delta;						//reset delta to whatever we want to achieve
			++j;

		}
	}

  // For fetching the last point of the last edge, the texStarts array is not
  // filled correctly, so we just iterate over its points to fetch the endpoint
  // correctly
  if(e_idx==n_edges-1){
    BundlingPoint current_point, last;
    int st = tex1Dfetch(texStarts, e_idx);
    while(1) {
      current_point = tex1Dfetch(texSites, ++st);
      if(current_point.x<0) break;
      last = current_point;
    }
	  *new_pts = last;
  }else{
	  *new_pts = tex1Dfetch(texSites,  tex1Dfetch(texStarts,e_idx+1)-2);
  }

	++omp;
  *new_pts++;

	if (omp<NP+1)
	{
	   *new_pts = *(new_pts-1);
	   ++new_pts;
 	   ++omp;
	}


	//!!if (crtd<delta)
	  //!!*new_pts++ = tex1Dfetch(texSites,i-1);

	if (omp!=NP+1)
		printf("***** Expected: %d, generated %d\n",NP+1,omp);

	new_pts->x = -1;							//add end-of-line marker

	state[threadIdx.x] = lState;						//update random generator
}






__global__ void kernelOffs(int* edges,int n_edges)
{
	//if (blockIdx.x==0 && threadIdx.x==0)
	{
		int cprev = edges[0];											//2. From 1, compute start-offset of resampled edges. Knowing this allows us to parallelize
		edges[0] = 0;													//   the resampling and writing the resampled points (in pass 3 below)
		for(int i=1;i<n_edges;++i)
		{
			int tmp = edges[i];
			edges[i] = edges[i-1] + cprev;
			cprev = tmp;
		}

		numControlPoints = edges[n_edges-1]+cprev;
	}
}



__global__ void kernelRandomInit(hiprandState* state)					//Initialize one random number generator state
{
	int id = threadIdx.x;
	hiprand_init(1234,id,0,state+id);
}

extern "C" void random_init(hiprandState* d_states)						//Initialize NTHREADS random generators.
{																		//We'll use them later in kernels when we need random numbers.
	kernelRandomInit<<<1,NTHREADS>>>(d_states);
	hipDeviceSynchronize();
}


extern "C" void resample(BundlingPoint* new_points,int& n_outpoints,int* out_edges,BundlingPoint* in_points,int n_inpoints,int* in_edges,int* h_edges,int n_edges,float delta,hiprandState* d_states,float jitter,
						 float* d_edgeProfile)
{
	int threads    = int(NTHREADS);										//Prepare the resample kernel
	int numedges_b = iAlignUp(n_edges,threads);							//Find higher multiple of blocksize than # edges
	int blocks     = int(numedges_b/threads);

	hipBindTexture(0,texSites,in_points);								//Bind 2D sites to a 1D texture
	hipBindTexture(0,texStarts,in_edges);								//Bind edge-start offsets in above vector to another 1D texture

	kernelResampleCount<<<blocks,threads>>>(out_edges,n_edges,delta);	//1. Compute #points that resampling produces on each edge. Store this in out_edges[]
	hipDeviceSynchronize();

	hipMemcpy(h_edges,out_edges,n_edges*sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	int cprev = h_edges[0];												//2. From 1, compute start-offset of resampled edges. Knowing this allows us to parallelize
	h_edges[0] = 0;														//   the resampling and writing the resampled points (in pass 3 below)
	for(int i=1;i<n_edges;++i)											//REMARK: This is the only still required CPU-GPU communication for the bundling algorithm..
	{
		int tmp = h_edges[i];
		h_edges[i] = h_edges[i-1] + cprev;
		cprev = tmp;
	}
	int NP = h_edges[n_edges-1]+cprev;
	h_edges[n_edges] = NP;												//Put an extra item at end, equal to the #points

	hipMemcpy(out_edges,h_edges,(n_edges+1)*sizeof(int),hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipBindTexture(0,texEdges,out_edges);											//3. Resample edges, producing same #points/edge as in pass 1. Put results in
	hipBindTexture(0,texEdgeProfile,d_edgeProfile);								//   contiguous vector 'new_points'
	kernelResample<<<blocks,threads>>>(new_points,n_edges,delta,jitter,d_states);
	hipDeviceSynchronize();

	hipUnbindTexture(texEdgeProfile);
	hipUnbindTexture(texEdges);
	hipUnbindTexture(texSites);
	hipUnbindTexture(texStarts);

	n_outpoints = NP;
}





__global__ void kernelDrawing2GL(GLBundlingPoint* out_points,int n_edges)
{
    int e_idx = IMAD(blockIdx.x, blockDim.x, threadIdx.x);              //Get edge number in texStarts
    if (e_idx>=n_edges) return;                                         //Careful not to index outside edge-vector

    int	offs    = tex1Dfetch(texStarts,e_idx);                          //Find idx in texSites of 1st edge-point on current edge

    out_points += offs;                                                 //Find position of 1st edge-point in texSites (and out_points too)

    GLBundlingPoint prev_point;

    for(;;++offs)
    {
        BundlingPoint   crtp = tex1Dfetch(texSites,offs);               //Current point on edge
        if (crtp.x<0)                                                   //End-of-edge marker?
        {                                                               //Simply copy previous point (always exists)
            *out_points = prev_point;
            break;
        }
        else                                                            //Not end-of-edge marker?
        {
            prev_point.coord  =  make_float2(crtp.x,crtp.y);                   //Remember it for next iteration
            prev_point.rgba[0] = 255;
            prev_point.rgba[1] = 128;
            prev_point.rgba[2] = 0;
            prev_point.rgba[3] = 255;


            *out_points = prev_point;                                   //Copy it to output
        }

        ++out_points;
    }
}




extern "C" void drawing2GL(GLBundlingPoint* gl_points, BundlingPoint* in_points, int n_inpoints, int* in_edges, int n_edges)
{
    int threads    = int(NTHREADS);										//Prepare the resample kernel
    int numedges_b = iAlignUp(n_edges,threads);							//Find higher multiple of blocksize than # edges
    int blocks     = int(numedges_b/threads);

    hipBindTexture(0,texSites,in_points);								//Bind 2D sites to a 1D texture
    hipBindTexture(0,texStarts,in_edges);								//Bind edge-start offsets in above vector to another 1D texture

    kernelDrawing2GL<<<blocks,threads>>>(gl_points,n_edges);

    hipDeviceSynchronize();

    hipUnbindTexture(texStarts);
    hipUnbindTexture(texSites);


//--------------------------------------------


}
